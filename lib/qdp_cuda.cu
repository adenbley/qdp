#include <iostream>

#include "qdp_cuda.h"

using namespace std;

namespace QDPCUDA {

  
  void getDeviceMem(void **mem , size_t size)
  {
    hipError_t ret;
    ret = hipMalloc(mem,size);
    cout << "hipMalloc     " << size << " : " << *mem << " " << string(hipGetErrorString(ret)) << endl;
  }
  void freeDeviceMem(void *mem)
  {
    hipError_t ret;
    ret = hipFree(mem);
    cout << "hipFree   : " << string(hipGetErrorString(ret)) << endl;
  }
  void getHostMem(void **mem , size_t size)
  {
    hipError_t ret;
    ret = hipHostAlloc(mem,size,hipHostMallocDefault);
    cout << "cudaHostMalloc " << size << " : " << string(hipGetErrorString(ret)) << endl;
  }
  void freeHostMem(void *mem)
  {
    hipError_t ret;
    ret = hipHostFree(mem);
    cout << "hipHostFree   : " << string(hipGetErrorString(ret)) << endl;
  }

  void copyToDevice(void *dest , void *src , size_t size)
  {
    hipError_t ret;
    ret = hipMemcpy(dest,src,size,hipMemcpyHostToDevice);
    cout << "hipMemcpy to device: " << string(hipGetErrorString(ret)) << endl;
  }
  void copyToHost(void *dest , void *src , size_t size)
  {
    hipError_t ret;
    ret = hipMemcpy(dest,src,size,hipMemcpyDeviceToHost);
    cout << "hipMemcpy to host: " << string(hipGetErrorString(ret)) << endl;
  }

}

