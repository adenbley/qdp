#include <iostream>

#include "qdp_cuda.h"

using namespace std;

namespace QDPCUDA {

  size_t total_device_memory = 0;
  
  void getDeviceMem(void **mem , size_t size)
  {
    hipError_t ret;
    ret = hipMalloc(mem,size);
#ifdef GPU_DEBUG
    cout << "hipMalloc     " << size << " : " << *mem << " " << string(hipGetErrorString(ret)) << endl;
#endif
    if (ret == hipSuccess)
      total_device_memory += size;
  }
  void freeDeviceMem(void *mem)
  {
    hipError_t ret;
    ret = hipFree(mem);
#ifdef GPU_DEBUG
    cout << "hipFree   : " << string(hipGetErrorString(ret)) << endl;
#endif
  }
  void getHostMem(void **mem , size_t size)
  {
    hipError_t ret;
    ret = hipHostAlloc(mem,size,hipHostMallocDefault);
#ifdef GPU_DEBUG
    cout << "cudaHostMalloc " << size << " : " << string(hipGetErrorString(ret)) << endl;
#endif
  }
  void freeHostMem(void *mem)
  {
    hipError_t ret;
    ret = hipHostFree(mem);
#ifdef GPU_DEBUG
    cout << "hipHostFree   : " << string(hipGetErrorString(ret)) << endl;
#endif
  }

  void copyToDevice(void *dest , void const *src , size_t size)
  {
    hipError_t ret;
    ret = hipMemcpy(dest,src,size,hipMemcpyHostToDevice);
#ifdef GPU_DEBUG
    cout << "hipMemcpy to device: " << string(hipGetErrorString(ret)) << endl;
#endif
  }
  void copyToHost(void *dest , void const *src , size_t size)
  {
    hipError_t ret;
    ret = hipMemcpy(dest,src,size,hipMemcpyDeviceToHost);
#ifdef GPU_DEBUG
    cout << "hipMemcpy to host: " << string(hipGetErrorString(ret)) << endl;
#endif
  }
  void copyHostToHost(void *dest , void const *src , size_t size)
  {
    hipError_t ret;
    ret = hipMemcpy(dest,src,size,hipMemcpyHostToHost);
#ifdef GPU_DEBUG
    cout << "hipMemcpy host to host: " << string(hipGetErrorString(ret)) << endl;
#endif
  }

}

