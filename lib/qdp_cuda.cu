#include <iostream>

#include "malloc.h"
#include "qdp_cuda.h"

using namespace std;

namespace QDPCUDA {


void inline cudp_check_error(std::string msg,hipError_t& ret)
{
#ifdef GPU_DEBUG
    cout << msg << endl;
    if (ret != hipSuccess) {
	cout << string(hipGetErrorString(ret)) << endl;
	exit(1);
    }
#else
    if (ret != hipSuccess) {
	cout << msg << endl;
	cout << string(hipGetErrorString(ret)) << endl;
	exit(1);
    }
#endif
}


  void hostRegister(void * ptr , size_t size, unsigned int flags)
  {
    hipError_t ret;
    ret = hipHostRegister(ptr, size, flags);
    cudp_check_error("hostRegister",ret);
  }
  
  void hostUnregister(void * ptr )
  {
    hipError_t ret;
    ret = hipHostUnregister(ptr);
    cudp_check_error("hostUnregister",ret);
  }
  
  void getDeviceMem(void **mem , size_t size)
  {
    hipError_t ret;
    ret = hipMalloc(mem,size);
    cudp_check_error("hipMalloc",ret);
  }

  void freeDeviceMem(void *mem)
  {
    hipError_t ret;
    ret = hipFree(mem);
    cudp_check_error("hipFree",ret);
  }

  void getHostMem(void **mem , size_t size)
  {
    hipError_t ret;
    ret = hipHostAlloc(mem,size,hipHostMallocDefault);
    cudp_check_error("cudaHostMalloc",ret);
  }

  void freeHostMem(void *mem)
  {
    hipError_t ret;
    ret = hipHostFree(mem);
    cudp_check_error("hipHostFree",ret);
  }

  void copyToDevice(void *dest , void const *src , size_t size)
  {
    hipError_t ret;
    ret = hipMemcpy(dest,src,size,hipMemcpyHostToDevice);
    cudp_check_error("hipMemcpy to device",ret);
  }

  void copyToHost(void *dest , void const *src , size_t size)
  {
    hipError_t ret;
    ret = hipMemcpy(dest,src,size,hipMemcpyDeviceToHost);
    cudp_check_error("hipMemcpy to host",ret);
  }

  void copyHostToHost(void *dest , void const *src , size_t size)
  {
    hipError_t ret;
    ret = hipMemcpy(dest,src,size,hipMemcpyHostToHost);
    cudp_check_error("hipMemcpy host to host",ret);
  }



}

