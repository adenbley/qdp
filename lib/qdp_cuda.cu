#include <iostream>

#include "qdp_cuda.h"

using namespace std;

namespace QDPCUDA {

  
  void getDeviceMem(void **mem , size_t size)
  {
    hipError_t ret;
    ret = hipMalloc(mem,size);
#ifdef GPU_DEBUG
    cout << "hipMalloc     " << size << " : " << *mem << " " << string(hipGetErrorString(ret)) << endl;
#endif
    if (ret != hipSuccess) {
      cout << "hipMalloc     " << size << " : " << *mem << " " << string(hipGetErrorString(ret)) << endl;
      exit(1);
    }
  }
  void freeDeviceMem(void *mem)
  {
    hipError_t ret;
    ret = hipFree(mem);
#ifdef GPU_DEBUG
    cout << "hipFree   : " << string(hipGetErrorString(ret)) << endl;
#endif
    if (ret != hipSuccess) {
      cout << "hipFree   : " << string(hipGetErrorString(ret)) << endl;
      exit(1);
    }
  }
  void getHostMem(void **mem , size_t size)
  {
    hipError_t ret;
    ret = hipHostAlloc(mem,size,hipHostMallocDefault);
#ifdef GPU_DEBUG
    cout << "cudaHostMalloc " << size << " : " << string(hipGetErrorString(ret)) << endl;
#endif
    if (ret != hipSuccess) {
      cout << "cudaHostMalloc " << size << " : " << string(hipGetErrorString(ret)) << endl;
      exit(1);
    }
  }
  void freeHostMem(void *mem)
  {
    hipError_t ret;
    ret = hipHostFree(mem);
#ifdef GPU_DEBUG
    cout << "hipHostFree   : " << string(hipGetErrorString(ret)) << endl;
#endif
    if (ret != hipSuccess) {
      cout << "hipHostFree   : " << string(hipGetErrorString(ret)) << endl;
      exit(1);
    }
  }

  void copyToDevice(void *dest , void const *src , size_t size)
  {
    hipError_t ret;
    ret = hipMemcpy(dest,src,size,hipMemcpyHostToDevice);
#ifdef GPU_DEBUG
    cout << "hipMemcpy to device: " << string(hipGetErrorString(ret)) << endl;
#endif
    if (ret != hipSuccess) {
      cout << "hipMemcpy to device: " << string(hipGetErrorString(ret)) << endl;
      exit(1);
    }
  }
  void copyToHost(void *dest , void const *src , size_t size)
  {
    hipError_t ret;
    ret = hipMemcpy(dest,src,size,hipMemcpyDeviceToHost);
#ifdef GPU_DEBUG
    cout << "hipMemcpy to host: " << string(hipGetErrorString(ret)) << endl;
#endif
    if (ret != hipSuccess) {
      cout << "hipMemcpy to host: " << string(hipGetErrorString(ret)) << endl;
      exit(1);
    }
  }
  void copyHostToHost(void *dest , void const *src , size_t size)
  {
    hipError_t ret;
    ret = hipMemcpy(dest,src,size,hipMemcpyHostToHost);
#ifdef GPU_DEBUG
    cout << "hipMemcpy host to host: " << string(hipGetErrorString(ret)) << endl;
#endif
    if (ret != hipSuccess) {
      cout << "hipMemcpy host to host: " << string(hipGetErrorString(ret)) << endl;
      exit(1);
    }
  }

}

